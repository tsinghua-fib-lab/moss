#include "utils/macro.h"
#include "utils/utils.cuh"

hipStream_t NewStream() {
  hipStream_t s;
  CUCHECK(hipStreamCreateWithFlags(&s, hipStreamNonBlocking));
  return s;
}
